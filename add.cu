
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

//Kernel function to add the elements of 2 arrays
__global__
void add(int n, float *x, float *y) {
	//Add in to go through array with parallel threads
	int index = threadIdx.x;
	int stride = blockDim.x;

	for (int i = index; i < n; i += stride)
		y[i] = x[i] + y[i];
}

int main(void) {
	int N = 1<<20;
	float *x, *y;

	//allocate unified memory - accessible from CPU or GPU
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	//initialize x and y arrays on the host
	for( int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
	
	//run kernel on 1 million elements on the GPU
	add<<<1, 256>>>(N, x, y);

	//wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	//check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	for( int i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(y[i]-3.0f));
	std::cout << "Max error: " << maxError << std::endl;

	//free memory
	hipFree(x);
	hipFree(y);

	return 0;
}
